#include "hip/hip_runtime.h"
/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"
#define TILE_WIDTH 16

// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	__shared__ float tileMs[TILE_WIDTH][TILE_WIDTH];
	__shared__ float tileNs[TILE_WIDTH][TILE_WIDTH];

	int tx = threadIdx.x; int ty = threadIdx.y;
	int bx = blockIdx.x; int by = blockIdx.y;

	// target element coordinates
	int row = by * TILE_WIDTH + ty;
	int column = bx * TILE_WIDTH + tx;

	float pValue = 0;

	// compute target element value
	for(int i=0;i<ceilf(M.width/(float)TILE_WIDTH);i++){
		// move the tiles and update shared memory value for new tile positions
		if(row < M.height && (i*TILE_WIDTH + tx)<M.width)
			tileMs[ty][tx] = M.elements[row*M.width + i*TILE_WIDTH + tx];
		else
			tileMs[ty][tx] = 0;
		if(column < N.width && (i*TILE_WIDTH + ty)<N.height)
			tileNs[ty][tx] = N.elements[(i*TILE_WIDTH + ty)*N.width + column];
		else
			tileNs[ty][tx] = 0;

		// after the entire tile's values are available, proceed
		__syncthreads();

		for(int j=0;j<TILE_WIDTH;j++)
			pValue += tileMs[ty][j] * tileNs[j][tx];
		// after the entire tile's values have been used, proceed
		__syncthreads();
	}
	// boundary check
	if(row < P.height && column < P.width)
		P.elements[row*P.width+column] = pValue;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
